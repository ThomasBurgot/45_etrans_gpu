#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

static const char *_cudaGetErrorEnum (hipfftResult error)
{
  switch (error)
    {
#define cr(x) case CUFFT_##x: return #x
      cr (SUCCESS);
      cr (INVALID_PLAN);
      cr (ALLOC_FAILED);
      cr (INVALID_TYPE);
      cr (INVALID_VALUE);
      cr (INTERNAL_ERROR);
      cr (EXEC_FAILED);
      cr (SETUP_FAILED);
      cr (INVALID_SIZE);
      cr (UNALIGNED_DATA);
#undef cr
    }

  return "UNKNOWN";
}

static inline void __cufftSafeCall (hipfftResult err, const char * file, const int line)
{
  if( HIPFFT_SUCCESS != err) 
    {
      fprintf (stderr, "CUFFT error at %s:%d\n", file, line);
      fprintf (stderr, "CUFFT error %d %s\n", err, _cudaGetErrorEnum (err)); 
      hipDeviceReset (); 
    }
 }

extern "C"
void
#ifdef TRANS_SINGLE
execute_plan_fftc_ (cufftHandle * PLANp, int * ISIGNp, cufftComplex * data)
#else
execute_plan_fftc_ (hipfftHandle * PLANp, int * ISIGNp, hipfftDoubleComplex * data)
#endif
{
  hipfftHandle plan = *PLANp;
  int ISIGN = *ISIGNp;
  
/*if (cudaDeviceSynchronize() != cudaSuccess){
  	fprintf(stderr, "Cuda error: Failed to synchronize\n");
  	return;	
}*/
  
  if (ISIGN== -1)
    {
#ifdef TRANS_SINGLE
    cufftSafeCall(cufftExecR2C(plan, (cufftReal*)data, data));
#else
    cufftSafeCall(hipfftExecD2Z(plan, (hipfftDoubleReal*)data, data));
#endif
    }
  else if (ISIGN== 1)
    {
#ifdef TRANS_SINGLE
    cufftSafeCall(cufftExecC2R(plan, data, (cufftReal*)data));
#else
    cufftSafeCall(hipfftExecZ2D(plan, data, (hipfftDoubleReal*)data));
#endif
    }
  else 
    {
      abort();
    }

// cudaDeviceSynchronize();

//if (cudaDeviceSynchronize() != cudaSuccess){
//	fprintf(stderr, "Cuda error: Failed to synchronize\n");
//	return;	
//}


}

