#define cufftSafeCall(err) __cufftSafeCall(err, __FILE__, __LINE__)

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

static const char *_cudaGetErrorEnum(hipfftResult error)
{
  switch (error)
    {
#define cr(x) case CUFFT_##x: return #x
      cr (SUCCESS);
      cr (INVALID_PLAN);
      cr (ALLOC_FAILED);
      cr (INVALID_TYPE);
      cr (INVALID_VALUE);
      cr (INTERNAL_ERROR);
      cr (EXEC_FAILED);
      cr (SETUP_FAILED);
      cr (INVALID_SIZE);
      cr (UNALIGNED_DATA);
#undef cr
    }
  return "<unknown>";
}

inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
{
  if( HIPFFT_SUCCESS != err) {
  fprintf(stderr, "CUFFT error at 1\n");
  fprintf(stderr, "CUFFT error in file '%s'\n",__FILE__);
  fprintf(stderr, "CUFFT error at 2\n");
  /*fprintf(stderr, "CUFFT error line '%s'\n",__LINE__);*/
  fprintf(stderr, "CUFFT error at 3\n");
  /*fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
  _cudaGetErrorEnum(err)); \*/
  fprintf(stderr, "CUFFT error %d: %s\nterminating!\n",err,_cudaGetErrorEnum(err)); \
  hipDeviceReset(); return; \
  }
}


static int allocatedWorkspace=0;
static void* planWorkspace;
static int planWorkspaceSize=100*1024*1024; //100MB
 
extern "C"
void
//create_plan_fftc_(cufftHandle *PLANp, int *ISIGNp, int *Np, int *LOTp, int *STRIDEp, int *DISTp)
create_plan_fftc_(hipfftHandle *PLANp, int *Np, int *ISTRIDEp, int *IDISTp, int *OSTRIDEp, int *ODISTp, int *ISIGNp, int *LOTp)
{
  int ISIGN = *ISIGNp;
  int N = *Np;
  int LOT = *LOTp;
  
  hipfftHandle plan;
  
  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return;	
    }
  
  
  // //create a single re-usable workspace
  // if(!allocatedWorkspace){
  //   allocatedWorkspace=1;
  //   //allocate plan workspace
  //   cudaMalloc(&planWorkspace,planWorkspaceSize);
  // }
  //
  // //disable auto allocation so we can re-use a single workspace (created above)
  //  cufftSetAutoAllocation(plan, false);
  
  int embed[1];
  int istride, ostride;
  int idist, odist;
  
  #ifdef TRANS_SINGLE
  cufftType cufft_1 = CUFFT_R2C;
  cufftType cufft_2 = CUFFT_C2R;
  #else
  hipfftType cufft_1 = HIPFFT_D2Z;
  hipfftType cufft_2 = HIPFFT_Z2D;
  #endif
  
  embed[0] = 1;
  istride   = *ISTRIDEp;
  idist     = *IDISTp;
  ostride   = *OSTRIDEp;
  odist     = *ODISTp;
  
  
  cufftSafeCall (hipfftCreate (&plan));
  
  if(1){
    printf("CreatePlan cuFFT\n","N=",N);
    printf("%s %d \n","plan=",plan);
    printf("%s %d \n","LOT=",LOT);
    printf("%s %d \n","ISIGN=",ISIGN);
    printf("%s %d \n","Np=",*Np);
    printf("%s %d \n","ISTRIDEp=",*ISTRIDEp);
    printf("%s %d \n","IDISTp=",*IDISTp);
    printf("%s %d \n","OSTRIDEp=",*OSTRIDEp);
    printf("%s %d \n","ODISTp=",*ODISTp);
    fflush (stdout);
  }
  
  
  hipfftType type = ISIGN < 0 ? cufft_1 : cufft_2;
  
  cufftSafeCall (hipfftPlanMany (&plan, 1, &N, embed, istride, idist, embed, ostride, odist, type, LOT));
  
  if (hipDeviceSynchronize() != hipSuccess)
    {
      fprintf(stderr, "Cuda error: Failed to synchronize\n");
      return;	
    }
  
  *PLANp=plan;
  
  // // get size used by this plan
  // size_t workSize;
  // cufftGetSize(plan,&workSize);
  //
  // // exit if we don't have enough space for the work area in the re-usable workspace
  // if(workSize > planWorkspaceSize){
  //   printf("create_plan_fftc: plan workspace size not large enough - exiting\n");
  // exit(1);
  // }
  
  
  return;
}

